#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <math.h>

__device__ float sigmoid(float z)
{
    float y = (float)1 / (1 + exp(-z));
    return y;
}

__global__ void elementMultDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_PCols) {
        int idx = col + row * num_PCols;
        d_P[idx] = d_M[idx] * d_N[idx];
    }
}

__global__ void activationFuncForwardDevice(float *d_Z, float *d_Y, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float z = d_Z[idx];

        d_Y[idx] = sigmoid(z);
    }
}

__global__ void activationFuncBackwardDevice(float *d_Z, float *d_dervA, float *d_dervZ, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float s = sigmoid(d_Z[idx]);
        
        d_dervZ[idx] =  d_dervA[idx] * s * (1  - s) ;
    }
}

// This is currently a non tiled version based on the text book implementation
__global__ void dotProductDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    // Row index of the P and M
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Col index of the P and N
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_NCols) {
        float pVal = 0.0;

        // Each thread computes one element of the block
        int i, j;
        for (i = 0, j = 0; i < num_NRows && j < num_MCols; i++, j++) {
            int m_idx = j + row * num_MCols;
            int n_idx = col + i * num_NCols;
            pVal += d_M[m_idx] * d_N[n_idx];
        }

        d_P[row * num_PCols + col] = pVal;
    }
}

// 
// Interface functions for the corresponding kernel functions.
//

// h_Y will ahve the output
void activationFuncForward(float *h_Z, float *h_Y, int numRows, int numCols)
{
    float *d_Z, *d_Y;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_Y, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_Y, h_Y, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncForwardDevice<<<gridDim, blockDim>>>(d_Z, d_Y, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_Y, d_Y, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_Y);
}

// h_dervZ will have the output
void activationFuncBackward(float *h_Z, float *h_dervA, float *h_dervZ, int numRows, int numCols)
{
    float *d_Z, *d_dervA, *d_dervZ;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_dervA, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMalloc((void**)&d_dervZ, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_dervA, h_dervA, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMemcpy(d_dervZ, h_dervZ, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncBackwardDevice<<<gridDim, blockDim>>>(d_Z, d_dervA, d_dervZ, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_dervZ, d_dervZ, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_dervA);
    hipFree(d_dervZ);
}

void dotProduct(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    if (num_MCols != num_NRows) {
        printf("(device) num_MCols != num_NRows\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    dotProductDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

void elementMult(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    if (num_MRows != num_NRows) {
        printf("(device) num_MRows!= num_NRows\n");
        exit(-1);
    }

    if (num_MCols != num_NCols) {
        printf("(device) num_MCols != num_NCols\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    elementMultDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

__global__ void scalarMultiplication(double scalar, double* M, int Rows, int Cols){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 

    if(r < Rows && c < Cols)
        M[r][c] *= scalar;
}




__global__ void updateWeights(float* d_w, float eta, float* d_dotP, float alpha, int Rows, int Cols){
    /*
        w -- set of weights being updated
        error -- the error by which the weights need to be updated
        layer -- can be the output-to-hidden layer OR the hidden-to-input layer
        alpha -- momentum
    */
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(r < Rows && c < Cols)
        d_w[r][c] = eta * d_dotP + alpha * d_w[r][c]

}
