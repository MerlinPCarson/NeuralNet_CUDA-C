#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <math.h>

__device__ float sigmoid(float z)
{
    float y = (float)1 / (1 + exp(-z));
    return y;
}

__global__ void elementMultDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_PCols) {
        int idx = col + row * num_PCols;
        d_P[idx] = d_M[idx] * d_N[idx];
    }
}

__global__ void batchPredsDevice(float * out_activations, unsigned short * batch, int output_size, int batch_size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < batch_size && col == 0)
    {
        unsigned short counter = 0;
        float maxValue = out_activations[row * output_size];
        for(int i = 1; i < output_size; ++i)
        {
            int idx = i + row*output_size;
            if(out_activations[idx] > maxValue)
            {
                maxValue = out_activations[idx];
                counter = i;
            }
        }
        batch[row] = counter;
    }
}

__global__ void activationFuncForwardDevice(float *d_Z, float *d_Y, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float z = d_Z[idx];

        d_Y[idx] = sigmoid(z);
    }
}

__global__ void activationFuncBackwardDevice(float *d_Z, float *d_dervA, float *d_dervZ, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float s = sigmoid(d_Z[idx]);
        
        d_dervZ[idx] =  d_dervA[idx] * s * (1  - s) ;
    }
}

__global__ void transposeDevice(float *d_M, float *d_N, int num_MRows, int num_MCols)
{
    int num_NRows = num_MCols;
    int num_NCols = num_MRows;

    // Row index of N
    int rowN = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Col index of N
    int colN = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Row index of M
    int rowM = colN;
    
    // Col index of M
    int colM = rowN;

    if (rowN < num_NRows && colN < num_NCols) {
        // Each thread computes one element of the block
        int n_idx = colN + rowN * num_NCols;
        int m_idx = colM + rowM * num_MCols;
        d_N[n_idx] = d_M[m_idx];
    }
}

// This is currently a non tiled version based on the text book implementation
__global__ void dotProductDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    // Row index of the P and M
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Col index of the P and N
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_NCols) {
        float pVal = 0.0;

        // Each thread computes one element of the block
        int i, j;
        for (i = 0, j = 0; i < num_NRows && j < num_MCols; i++, j++) {
            int m_idx = j + row * num_MCols;
            int n_idx = col + i * num_NCols;
            pVal += d_M[m_idx] * d_N[n_idx];
        }

        d_P[row * num_PCols + col] = pVal;
    }
}

// d_T is 1D (batchSize), d_O is 2D (batchSize, numLabels)
// numRows = batch size
// d_sampleSquareErr: array to store the square error for each sample
__global__ void mseDevice(
    unsigned short *d_T,
    float *d_O,
    float *d_sampleSquareErr,
    float *batchLoss,
    int batchSize,
    int numLabels
    )
{
    int batchId = blockIdx.x * blockDim.x + threadIdx.x;
    if(batchId < batchSize){
      int t_idx = d_T[batchId];
      //printf("batch ID: %d, target: %f\n", batchId, d_T[batchId]);
  
      *batchLoss = 0;
      
      // Sanity check
      if (t_idx >= numLabels) {
          printf("t_idx (%d) >= numLabels (%d)\n", t_idx, numLabels);
          return;
      }
  
      // Now go through each of the output values and calculate the MSE
      float err = 0;
      for (int j = 0; j < numLabels; j++) {
          int o_idx = j + batchId * numLabels;
  
          if (t_idx == j) {
              // If this is the same as the expected output
              float diff = 1 - d_O[o_idx];
//              printf("diff: %f", diff);
              err += diff * diff;
          }
          else {
              float diff = d_O[o_idx];
//              printf("diff: %f", diff);
              err += diff * diff;
          }
      }
      //printf("err: %f\n", err);
      d_sampleSquareErr[batchId] = err;
    } 
    __syncthreads();

    // Calculate the square error for the batch
    
    // Need only one thread to do this
    if (batchId == 0) {
        for (int i = 0; i < batchSize; i++) {
            *batchLoss += d_sampleSquareErr[i];
        }
        *batchLoss /= (float)2;
        *batchLoss /= (float)batchSize;
//        printf("batch err: %f\n", *batchLoss);
    }
}

// 
// Interface functions for the corresponding kernel functions.
//

// h_T is 1D (batchSize), h_O is 2D (batchSize, numLabels)
// numRows = batch size
float MSE(unsigned short *h_T, float *h_O, int batchSize, int numLabels)
{
    float h_batchLoss = 0;
    unsigned short *d_T;
    float *d_O, *d_sampleSquareErr, *d_batchLoss;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_T, batchSize * sizeof(unsigned short));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_O, batchSize * numLabels * sizeof(float));
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMalloc((void**)&d_sampleSquareErr, batchSize * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_batchLoss, sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_T, h_T, batchSize * sizeof(unsigned short), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_O, h_O, batchSize * numLabels * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)batchSize / BLOCK_WIDTH), 1, 1);
    dim3 blockDim(BLOCK_WIDTH, 1, 1);

    //printf("Activations:\n");
    //printMatrix(h_O, batchSize, numLabels);
    // Call the kernel
    mseDevice<<<gridDim, blockDim>>>(d_T, d_O, d_sampleSquareErr, d_batchLoss, batchSize, numLabels);

    // Copy back to host
    cudaStatus = hipMemcpy(&h_batchLoss, d_batchLoss, sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_T);
    hipFree(d_O);
    hipFree(d_sampleSquareErr);
    hipFree(d_batchLoss);
    
    return h_batchLoss;
}

// h_Y will have the output
void activationFuncForward(float *h_Z, float *h_Y, int numRows, int numCols)
{
    float *d_Z, *d_Y;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_Y, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_Y, h_Y, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncForwardDevice<<<gridDim, blockDim>>>(d_Z, d_Y, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_Y, d_Y, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_Y);
}

// h_dervZ will have the output
void activationFuncBackward(float *h_Z, float *h_dervA, float *h_dervZ, int numRows, int numCols)
{
    float *d_Z, *d_dervA, *d_dervZ;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_dervA, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMalloc((void**)&d_dervZ, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_dervA, h_dervA, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMemcpy(d_dervZ, h_dervZ, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncBackwardDevice<<<gridDim, blockDim>>>(d_Z, d_dervA, d_dervZ, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_dervZ, d_dervZ, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_dervA);
    hipFree(d_dervZ);
}

void dotProduct(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    if (num_MCols != num_NRows) {
        printf("(device) num_MCols != num_NRows\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    dotProductDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

// The transposed matrix h_N will have rows = num_MCols, cols = num_MRows
void transpose(float *h_M, float *h_N, int num_MRows, int num_MCols)
{
    float *d_M, *d_N;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    // The rows and cols are interchanged here because of the transpose
    dim3 gridDim((int)ceil((float)num_MRows / BLOCK_WIDTH), (int)ceil((float)num_MCols / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    transposeDevice<<<gridDim, blockDim>>>(d_M, d_N, num_MRows, num_MCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_N, d_N, num_MRows * num_MCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
}

void batchPreds(float * h_activations, unsigned short * h_batchPreds, int activation_size, int b_size)
{
    float *d_activations;
    unsigned short *d_batchPreds;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_activations, activation_size* b_size* sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_batchPreds, activation_size * sizeof(unsigned short));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_activations, h_activations, activation_size * b_size * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_batchPreds, h_batchPreds, activation_size * sizeof(unsigned short), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)activation_size / BLOCK_WIDTH), (int)ceil((float) b_size / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    batchPredsDevice<<<gridDim, blockDim>>>(d_activations, d_batchPreds, activation_size, b_size);

    //copy back to host
    cudaStatus = hipMemcpy(h_batchPreds, d_batchPreds, activation_size * sizeof(unsigned short), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    hipFree(d_activations);
    hipFree(d_batchPreds);

}

void elementMult(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    if (num_MRows != num_NRows) {
        printf("(device) num_MRows!= num_NRows\n");
        exit(-1);
    }

    if (num_MCols != num_NCols) {
        printf("(device) num_MCols != num_NCols\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    elementMultDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

__global__ void scalarMultiplication(double scalar, double* M, int Rows, int Cols){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 

    if(r < Rows && c < Cols)
        M[r*Cols + c] *= scalar;
}




__global__ void updateWeights(float eta, float alpha, float* d_dotP, int Rows, int Cols, float* d_w){
    /*
        w -- set of weights being updated
        error -- the error by which the weights need to be updated
        layer -- can be the output-to-hidden layer OR the hidden-to-input layer
        alpha -- momentum
    */

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(r < Rows && c < Cols){
        int index = r*Cols + c;
        d_w[index] += eta * d_dotP[index]/BATCH_SIZE;// + alpha * d_w[index];
    }

}

__global__ void outputError(float* d_error, unsigned short* t, float* d_out_layer, int Rows, int Cols){
    /*
        d_error   -- delta_k
        targets    -- one hot encode 1D array containing 0.9 for target label
        d_out_layer -- the squashed activations for the output layer
        Rows      -- should be 1 as they are all 1D arrays
        Cols      -- should be the number of ouput nodes 
    */
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(r < Rows && c < Cols){ 
        int index = r*Cols + c;
        //printf("target: %hu, index: %d\n", t[r], c);
        if(t[r] == c)
            // 2x10               2x10                    2x10                1        2x10
            d_error[index] = d_out_layer[index] * (1 - d_out_layer[index]) * (1 - d_out_layer[index]);
        else 
            d_error[index] = d_out_layer[index] * (1 - d_out_layer[index]) * (0 - d_out_layer[index]);
    }
    
}


__global__ void hiddenError(float* d_error, float* d_dotP, float* d_hidden_layer, int Rows, int Cols){
    /*
    d_error         -- delta_j    
    d_dotP          -- the output error dot output weights
    d_hidden_layer  -- the hidden activations
    Rows            -- should be 1 as they are all 1D arrays
    Cols            -- should be the number of ouput nodes 
    */
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(r < Rows && c < Cols){
        int index = r*Cols + c;
        // 2x10               2x10                      2x10                     2x10
        d_error[index] = d_hidden_layer[index] * (1 - d_hidden_layer[index]) * (d_dotP[index]);
    }

}




void error_function(unsigned short * t, float* z, float* h, float* output_weights, float* delta_k, float* delta_j){
    
    //--------------  DEEIVCE Prep ----------------------
  float *d_z, *d_h, *d_k, *d_j;
  unsigned short *d_t;
  float *dotP, *d_dotP; 
  int outRows    = BATCH_SIZE,  outCols    = NUM_LABELS;
  int hiddenRows = BATCH_SIZE,  hiddenCols = HIDDEN_SIZE;
  
  
  hipError_t cudaStatus;
  cudaStatus = hipMalloc((void**)&d_t, BATCH_SIZE * sizeof(unsigned short));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_t, t, BATCH_SIZE * sizeof(unsigned short), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  cudaStatus = hipMalloc((void**)&d_z, outRows * outCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_z, z, outRows * outCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  cudaStatus = hipMalloc((void**)&d_h, hiddenRows * hiddenCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_h, h, hiddenRows * hiddenCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  
  cudaStatus = hipMalloc((void**)&d_k, outRows * outCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_k, delta_k, outRows * outCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  cudaStatus = hipMalloc((void**)&d_j, hiddenRows * hiddenCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_j, delta_j, hiddenRows * hiddenCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
    
  
  // call kernel for weight update for each thread to update a weight
  int blockX = ceil((float)outRows/BLOCK_WIDTH);
  int blockY = ceil((float)outCols/BLOCK_WIDTH);
  int threadX = BLOCK_WIDTH;
  int threadY = BLOCK_WIDTH;
  dim3 dimGrid(blockX,   blockY,  1);
  dim3 dimBlock(threadX, threadY, 1);
  //--------------  END: DEEIVCE Prep  ----------------------

//  for(int i=0; i < BATCH_SIZE; ++i){
//    printf("target: %hu ", t[i]);  
//  }

  outputError<<<dimGrid, dimBlock>>>(d_k, d_t, d_z, outRows, outCols ); 

  
  // copy back to the host because we need delta K for the dotP
  cudaStatus = hipMemcpy(delta_k, d_k, BATCH_SIZE * outCols * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError(cudaStatus);

//  printf("Delta K\n");
//  printMatrix(delta_k, outRows, outCols);
//  printf("\n");

  int delta_kRows = outRows;
  int delta_kCols = outCols;
  

  float* errorTransposed;
  errorTransposed = (float*)malloc(outRows*outCols*sizeof(float));
  transpose(delta_k, errorTransposed, outRows, outCols);
  
  int pRows = HIDDEN_SIZE,   pCols = BATCH_SIZE;
  dotP = (float*)malloc(pRows*pCols*sizeof(float));
  //     output weights    dot  output error Transposed = dotP
  //                2x10    @      10x1   = 2x1
  //HIDDEN_SIZE x NUM_LABEL @  NUM_LABEL x BATCH_SIZE  = HIDDEN_SIZE x BATCHSIZE
  //dotProduct((float*)output_weights, errorTransposed, dotP, HIDDEN_SIZE, NUM_LABELS, delta_kCols, delta_kRows);
  //printf("delta_kCols %d, delta_kRows %d", delta_kCols, delta_kRows);
  dotProduct((float*)output_weights, errorTransposed, dotP, HIDDEN_SIZE, NUM_LABELS, NUM_LABELS, BATCH_SIZE);
 

//  printf("Delta Ja\n");
//  printMatrix(dotP, HIDDEN_SIZE, BATCH_SIZE);
//  printf("\n");

  
  
  // Prep for hidden error
  blockX = ceil((float)hiddenCols/BLOCK_WIDTH);
  blockY = ceil((float)hiddenRows/BLOCK_WIDTH);
  threadX = BLOCK_WIDTH;
  threadY = BLOCK_WIDTH;
  dim3 dimGrid2(blockX,   blockY,  1);
  dim3 dimBlock2(threadX, threadY, 1);
  
  
  // used for the dot product of output error and output weights
  cudaStatus = hipMalloc((void**)&d_dotP, pRows * pCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_dotP, dotP, pRows * pCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  

  hiddenError<<<dimGrid2, dimBlock2>>>(d_j, d_dotP, d_h, hiddenRows, hiddenCols );
  
//  printf("hidden activations\n");
//  printMatrix(h, BATCH_SIZE, HIDDEN_SIZE);
//  printf("\n");

  // copy back to the host variables
  cudaStatus = hipMemcpy(delta_j, d_j, hiddenRows * hiddenCols * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError(cudaStatus);
  
//  printf("Delta J\n");
//  printMatrix(delta_j, hiddenRows, hiddenCols);
//  printf("\n");


  // deallocate device memory
  hipFree(d_z);
  hipFree(d_h);
  hipFree(d_k);
  hipFree(d_j);
  hipFree(d_dotP);

  
  free(errorTransposed);
  free(dotP);
}


void update_weights(float eta, float alpha, float* weights, int wRows, int wCols, float* dotP, int pRows, int pCols){
/*
    dotP -- error Transposed @ current layer activations
*/

  //--------------  DEEIVCE Prep ----------------------
  float *d_w,  *d_dotP;

  hipError_t cudaStatus;    
  cudaStatus = hipMalloc((void**)&d_w, wRows * wCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_w, weights, wRows * wCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);

  cudaStatus = hipMalloc((void**)&d_dotP, pRows * pCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(  d_dotP,  dotP, pRows * pCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);


  // call kernel for weight update for each thread to update a weight
  int blockX = ceil((float)wCols / BLOCK_WIDTH);
  int blockY = ceil((float)wRows / BLOCK_WIDTH);
  int threadX = BLOCK_WIDTH;
  int threadY = BLOCK_WIDTH;
  dim3 dimGrid(blockX,   blockY,  1);
  dim3 dimBlock(threadX, threadY, 1);
  //--------------  END: DEEIVCE Prep ----------------------

                          
  // output-hidden    (1x10) hidden activations  DOT  error(1x10)
  // hidden-input     (1x785) inputs  DOT  error(1x10) 
//  if(wRows == HIDDEN_SIZE){
//    printf("pre update\n");
//    printMatrix(weights, wRows, wCols);
//
//    printf("errors\n");
//    printMatrix(dotP, wRows, wCols);
//  }
  updateWeights<<<dimGrid, dimBlock>>>(eta, alpha, d_dotP, wRows, wCols, d_w);

  
    // copy back to the host variables
  cudaStatus = hipMemcpy(weights, d_w,  wRows * wCols * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError(cudaStatus);
  
  //if(wRows == HIDDEN_SIZE){
  //  printf("post update\n");
  //  printMatrix(weights, wRows, wCols);
  //}
    
    // deallocate device memory
  hipFree(d_w);
  hipFree(d_dotP);
}
