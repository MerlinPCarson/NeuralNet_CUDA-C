#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <math.h>

__device__ float sigmoid(float z)
{
    float y = (float)1 / (1 + exp(-z));
    return y;
}

__global__ void elementMultDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_PCols) {
        int idx = col + row * num_PCols;
        d_P[idx] = d_M[idx] * d_N[idx];
    }
}

__global__ void batchPredsDevice(float * out_activations, int * batch, int output_size, int batch_size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < batch_size && col == 0)
    {
        int counter = 0;
        for(int i = 1; i < output_size; ++i)
        {
            int idx = i + row*output_size;
            if(out_activations[idx] > out_activations[idx-1])
            {
                counter = i;
            }
        }
        batch[row] = counter;
    }
}

__global__ void activationFuncForwardDevice(float *d_Z, float *d_Y, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float z = d_Z[idx];

        d_Y[idx] = sigmoid(z);
    }
}

__global__ void activationFuncBackwardDevice(float *d_Z, float *d_dervA, float *d_dervZ, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float s = sigmoid(d_Z[idx]);
        
        d_dervZ[idx] =  d_dervA[idx] * s * (1  - s) ;
    }
}

__global__ void transposeDevice(float *d_M, float *d_N, int num_MRows, int num_MCols)
{
    int num_NRows = num_MCols;
    int num_NCols = num_MRows;

    // Row index of N
    int rowN = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Col index of N
    int colN = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Row index of M
    int rowM = colN;
    
    // Col index of M
    int colM = rowN;

    if (rowN < num_NRows && colN < num_NCols) {
        // Each thread computes one element of the block
        int n_idx = colN + rowN * num_NCols;
        int m_idx = colM + rowM * num_MCols;
        d_N[n_idx] = d_M[m_idx];
    }
}

// This is currently a non tiled version based on the text book implementation
__global__ void dotProductDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    // Row index of the P and M
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Col index of the P and N
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_NCols) {
        float pVal = 0.0;

        // Each thread computes one element of the block
        int i, j;
        for (i = 0, j = 0; i < num_NRows && j < num_MCols; i++, j++) {
            int m_idx = j + row * num_MCols;
            int n_idx = col + i * num_NCols;
            pVal += d_M[m_idx] * d_N[n_idx];
        }

        d_P[row * num_PCols + col] = pVal;
    }
}

// d_T is 1D (batchSize), d_O is 2D (batchSize, numLabels)
// numRows = batch size
// d_sampleSquareErr: array to store the square error for each sample
__global__ void mseDevice(
    float *d_T,
    float *d_O,
    float *d_sampleSquareErr,
    float *batchLoss,
    int batchSize,
    int numLabels
    )
{
    int batchId = blockIdx.x * blockDim.x + threadIdx.x;
    int t_idx = d_T[batchId];

    *batchLoss = 0;
    
    // Sanity check
    if (t_idx >= numLabels) {
        printf("t_idx (%d) >= numLabels (%d)\n", t_idx, numLabels);
        return;
    }

    // Now go through each of the output values and calculate the MSE
    float err = 0;
    for (int j = 0; j < numLabels; j++) {
        int o_idx = j + batchId * numLabels;

        if (t_idx == j) {
            // If this is the same as the expected output
            float diff = 1 - d_O[o_idx];
            err += diff * diff;
        }
        else {
            float diff = d_O[o_idx];
            err += diff * diff;
        }
    }
    d_sampleSquareErr[batchId] = err;
    
    __syncthreads();

    // Calculate the square error for the batch
    
    // Need only one thread to do this
    if (batchId == 0) {
        for (int i = 0; i < batchSize; i++) {
            *batchLoss += d_sampleSquareErr[i];
        }
        *batchLoss /= (float)2;
        *batchLoss /= (float)batchSize;
    }
}

// 
// Interface functions for the corresponding kernel functions.
//

// h_T is 1D (batchSize), h_O is 2D (batchSize, numLabels)
// numRows = batch size
float MSE(float *h_T, float *h_O, int batchSize, int numLabels)
{
    float h_batchLoss = 0;
    float *d_T, *d_O, *d_sampleSquareErr, *d_batchLoss;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_T, batchSize * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_O, batchSize * numLabels * sizeof(float));
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMalloc((void**)&d_sampleSquareErr, batchSize * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_batchLoss, sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_T, h_T, batchSize * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_O, h_O, batchSize * numLabels * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)batchSize / BLOCK_WIDTH), 1, 1);
    dim3 blockDim(BLOCK_WIDTH, 1, 1);

    // Call the kernel
    mseDevice<<<gridDim, blockDim>>>(d_T, d_O, d_sampleSquareErr, d_batchLoss, batchSize, numLabels);

    // Copy back to host
    cudaStatus = hipMemcpy(&h_batchLoss, d_batchLoss, sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_T);
    hipFree(d_O);
    hipFree(d_sampleSquareErr);
    hipFree(d_batchLoss);
    
    return h_batchLoss;
}

// h_Y will have the output
void activationFuncForward(float *h_Z, float *h_Y, int numRows, int numCols)
{
    float *d_Z, *d_Y;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_Y, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_Y, h_Y, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncForwardDevice<<<gridDim, blockDim>>>(d_Z, d_Y, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_Y, d_Y, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_Y);
}

// h_dervZ will have the output
void activationFuncBackward(float *h_Z, float *h_dervA, float *h_dervZ, int numRows, int numCols)
{
    float *d_Z, *d_dervA, *d_dervZ;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_dervA, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMalloc((void**)&d_dervZ, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_dervA, h_dervA, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMemcpy(d_dervZ, h_dervZ, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncBackwardDevice<<<gridDim, blockDim>>>(d_Z, d_dervA, d_dervZ, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_dervZ, d_dervZ, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_dervA);
    hipFree(d_dervZ);
}

void dotProduct(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    if (num_MCols != num_NRows) {
        printf("(device) num_MCols != num_NRows\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    dotProductDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

// The transposed matrix h_N will have rows = num_MCols, cols = num_MRows
void transpose(float *h_M, float *h_N, int num_MRows, int num_MCols)
{
    float *d_M, *d_N;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    // The rows and cols are interchanged here because of the transpose
    dim3 gridDim((int)ceil((float)num_MRows / BLOCK_WIDTH), (int)ceil((float)num_MCols / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    transposeDevice<<<gridDim, blockDim>>>(d_M, d_N, num_MRows, num_MCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_N, d_N, num_MRows * num_MCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
}

void batchPreds(float * h_activations, int * h_batchPreds, int activation_size, int b_size)
{
    float *d_activations;
    int *d_batchPreds;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_activations, activation_size* b_size* sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_batchPreds, activation_size * sizeof(int));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_activations, h_activations, activation_size * b_size * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_batchPreds, h_batchPreds, activation_size * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)activation_size / BLOCK_WIDTH), (int)ceil((float) b_size / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    batchPredsDevice<<<gridDim, blockDim>>>(d_activations, d_batchPreds, activation_size, b_size);

    //copy back to host
    cudaStatus = hipMemcpy(h_batchPreds, d_batchPreds, activation_size * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    hipFree(d_activations);
    hipFree(d_batchPreds);

}

void elementMult(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    if (num_MRows != num_NRows) {
        printf("(device) num_MRows!= num_NRows\n");
        exit(-1);
    }

    if (num_MCols != num_NCols) {
        printf("(device) num_MCols != num_NCols\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    elementMultDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}
