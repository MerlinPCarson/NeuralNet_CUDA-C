#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <math.h>


// This is currently a non tiled version based on the text book implementation
__global__ void dotProductDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    // Row index of the P and M
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Col index of the P and N
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_NCols) {
        float pVal = 0.0;

        // Each thread computes one element of the block
        int i, j;
        for (i = 0, j = 0; i < num_NRows && j < num_MCols; i++, j++) {
            int m_idx = j + row * num_MCols;
            int n_idx = col + i * num_NCols;
            pVal += d_M[m_idx] * d_N[n_idx];
        }

        d_P[row * num_PCols + col] = pVal;
    }
}

void dotProduct(float* d_M, float* d_N, float* d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    if (num_MCols != num_NRows) {
        printf("(device) num_MCols != num_NRows\n");
        exit(-1);
    }

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    dotProductDevice << <gridDim, blockDim >> > (d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);
}
