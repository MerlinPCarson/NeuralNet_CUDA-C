#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#define DEBUG

__device__ float sigmoid(float z)
{
    float y = (float)1 / (1 + exp(-z));
    return y;
}

__global__ void elementMultDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_PCols) {
        int idx = col + row * num_PCols;
        d_P[idx] = d_M[idx] * d_N[idx];
    }
}

__global__ void batchPredsDevice(float * out_activations, int * batch, int output_size, int batch_size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < batch_size && col == 0)
    {
        int counter = 0;
        float maxValue = out_activations[row * output_size];
        for(int i = 1; i < output_size; ++i)
        {
            int idx = i + row*output_size;
            if(out_activations[idx] > maxValue)
            {
                maxValue = out_activations[idx];
                counter = i;
            }
        }
        batch[row] = counter;
    }
}

__global__ void activationFuncForwardDevice(float *d_Z, float *d_Y, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float z = d_Z[idx];

        d_Y[idx] = sigmoid(z);
    }
}

__global__ void activationFuncBackwardDevice(float *d_Z, float *d_dervA, float *d_dervZ, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = col + row * numCols;
        float s = sigmoid(d_Z[idx]);
        
        d_dervZ[idx] =  d_dervA[idx] * s * (1  - s) ;
    }
}

__global__ void transposeDevice(float *d_M, float *d_N, int num_MRows, int num_MCols)
{
    int num_NRows = num_MCols;
    int num_NCols = num_MRows;

    // Row index of N
    int rowN = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Col index of N
    int colN = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Row index of M
    int rowM = colN;
    
    // Col index of M
    int colM = rowN;

    if (rowN < num_NRows && colN < num_NCols) {
        // Each thread computes one element of the block
        int n_idx = colN + rowN * num_NCols;
        int m_idx = colM + rowM * num_MCols;
        d_N[n_idx] = d_M[m_idx];
    }
}

// This is currently a non tiled version based on the text book implementation
__global__ void dotProductDevice(float *d_M, float *d_N, float *d_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    // Row index of the P and M
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Col index of the P and N
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_PRows && col < num_NCols) {
        float pVal = 0.0;

        // Each thread computes one element of the block
        int i, j;
        for (i = 0, j = 0; i < num_NRows && j < num_MCols; i++, j++) {
            int m_idx = j + row * num_MCols;
            int n_idx = col + i * num_NCols;
            pVal += d_M[m_idx] * d_N[n_idx];
        }

        d_P[row * num_PCols + col] = pVal;
    }
}

// d_T is 1D (batchSize), d_O is 2D (batchSize, numLabels)
// numRows = batch size
// d_sampleSquareErr: array to store the square error for each sample
__global__ void mseDevice(
    float *d_T,
    float *d_O,
    float *d_sampleSquareErr,
    float *batchLoss,
    int batchSize,
    int numLabels
    )
{
    int batchId = blockIdx.x * blockDim.x + threadIdx.x;
    int t_idx = d_T[batchId];

    *batchLoss = 0;
    
    // Sanity check
    if (t_idx >= numLabels) {
        printf("t_idx (%d) >= numLabels (%d)\n", t_idx, numLabels);
        return;
    }

    // Now go through each of the output values and calculate the MSE
    float err = 0;
    for (int j = 0; j < numLabels; j++) {
        int o_idx = j + batchId * numLabels;

        if (t_idx == j) {
            // If this is the same as the expected output
            float diff = 1 - d_O[o_idx];
            err += diff * diff;
        }
        else {
            float diff = d_O[o_idx];
            err += diff * diff;
        }
    }
    d_sampleSquareErr[batchId] = err;
    
    __syncthreads();

    // Calculate the square error for the batch
    
    // Need only one thread to do this
    if (batchId == 0) {
        for (int i = 0; i < batchSize; i++) {
            *batchLoss += d_sampleSquareErr[i];
        }
        *batchLoss /= (float)2;
        *batchLoss /= (float)batchSize;
    }
}

// 
// Interface functions for the corresponding kernel functions.
//

// h_T is 1D (batchSize), h_O is 2D (batchSize, numLabels)
// numRows = batch size
float MSE(float *h_T, float *h_O, int batchSize, int numLabels)
{
    float h_batchLoss = 0;
    float *d_T, *d_O, *d_sampleSquareErr, *d_batchLoss;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_T, batchSize * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_O, batchSize * numLabels * sizeof(float));
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMalloc((void**)&d_sampleSquareErr, batchSize * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_batchLoss, sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_T, h_T, batchSize * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_O, h_O, batchSize * numLabels * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)batchSize / BLOCK_WIDTH), 1, 1);
    dim3 blockDim(BLOCK_WIDTH, 1, 1);

    // Call the kernel
    mseDevice<<<gridDim, blockDim>>>(d_T, d_O, d_sampleSquareErr, d_batchLoss, batchSize, numLabels);

    // Copy back to host
    cudaStatus = hipMemcpy(&h_batchLoss, d_batchLoss, sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_T);
    hipFree(d_O);
    hipFree(d_sampleSquareErr);
    hipFree(d_batchLoss);
    
    return h_batchLoss;
}

// h_Y will have the output
void activationFuncForward(float *h_Z, float *h_Y, int numRows, int numCols)
{
    float *d_Z, *d_Y;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_Y, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_Y, h_Y, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncForwardDevice<<<gridDim, blockDim>>>(d_Z, d_Y, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_Y, d_Y, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_Y);
}

// h_dervZ will have the output
void activationFuncBackward(float *h_Z, float *h_dervA, float *h_dervZ, int numRows, int numCols)
{
    float *d_Z, *d_dervA, *d_dervZ;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_Z, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_dervA, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMalloc((void**)&d_dervZ, numRows * numCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_Z, h_Z, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_dervA, h_dervA, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    cudaStatus = hipMemcpy(d_dervZ, h_dervZ, numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    dim3 gridDim((int)ceil((float)numCols / BLOCK_WIDTH), (int)ceil((float)numRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    activationFuncBackwardDevice<<<gridDim, blockDim>>>(d_Z, d_dervA, d_dervZ, numRows, numCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_dervZ, d_dervZ, numRows * numCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_Z);
    hipFree(d_dervA);
    hipFree(d_dervZ);
}

void dotProduct(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_NCols;

    if (num_MCols != num_NRows) {
        printf("(device) num_MCols != num_NRows\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    dotProductDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

// The transposed matrix h_N will have rows = num_MCols, cols = num_MRows
void transpose(float *h_M, float *h_N, int num_MRows, int num_MCols)
{
    float *d_M, *d_N;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);
    
    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);
    
    // The rows and cols are interchanged here because of the transpose
    dim3 gridDim((int)ceil((float)num_MRows / BLOCK_WIDTH), (int)ceil((float)num_MCols / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    transposeDevice<<<gridDim, blockDim>>>(d_M, d_N, num_MRows, num_MCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_N, d_N, num_MRows * num_MCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
}

void batchPreds(float * h_activations, int * h_batchPreds, int activation_size, int b_size)
{
    float *d_activations;
    int *d_batchPreds;
    hipError_t cudaStatus;

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_activations, activation_size* b_size* sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_batchPreds, activation_size * sizeof(int));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_activations, h_activations, activation_size * b_size * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_batchPreds, h_batchPreds, activation_size * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)activation_size / BLOCK_WIDTH), (int)ceil((float) b_size / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    batchPredsDevice<<<gridDim, blockDim>>>(d_activations, d_batchPreds, activation_size, b_size);

    //copy back to host
    cudaStatus = hipMemcpy(h_batchPreds, d_batchPreds, activation_size * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    hipFree(d_activations);
    hipFree(d_batchPreds);

}

void elementMult(float *h_M, float *h_N, float *h_P, int num_MRows, int num_MCols, int num_NRows, int num_NCols)
{
    float *d_M, *d_N, *d_P;
    hipError_t cudaStatus;
    int num_PRows = num_MRows;
    int num_PCols = num_MCols;

    if (num_MRows != num_NRows) {
        printf("(device) num_MRows!= num_NRows\n");
        exit(-1);
    }

    if (num_MCols != num_NCols) {
        printf("(device) num_MCols != num_NCols\n");
        exit(-1);
    }

    // Allocate memory for device variables
    cudaStatus = hipMalloc((void**)&d_M, num_MRows * num_MCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_N, num_NRows * num_NCols * sizeof(float));
    cudaCheckError(cudaStatus);

    cudaStatus = hipMalloc((void**)&d_P, num_PRows * num_PCols * sizeof(float));
    cudaCheckError(cudaStatus);

    // Copy data to GPU
    cudaStatus = hipMemcpy(d_M, h_M, num_MRows * num_MCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_N, h_N, num_NRows * num_NCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    cudaStatus = hipMemcpy(d_P, h_P, num_PRows * num_PCols * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(cudaStatus);

    dim3 gridDim((int)ceil((float)num_PCols / BLOCK_WIDTH), (int)ceil((float)num_PRows / BLOCK_WIDTH), 1);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Call the kernel
    elementMultDevice<<<gridDim, blockDim>>>(d_M, d_N, d_P, num_MRows, num_MCols, num_NRows, num_NCols);

    // Copy back to host
    cudaStatus = hipMemcpy(h_P, d_P, num_PRows * num_PCols * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(cudaStatus);

    // Free device memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

__global__ void scalarMultiplication(double scalar, double* M, int Rows, int Cols){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 

    if(r < Rows && c < Cols)
        M[r*Cols + c] *= scalar;
}




__global__ void updateWeights(float eta, float alpha, float* d_dotP, int Rows, int Cols, float* d_w){
    /*
        w -- set of weights being updated
        error -- the error by which the weights need to be updated
        layer -- can be the output-to-hidden layer OR the hidden-to-input layer
        alpha -- momentum
    */

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(r < Rows && c < Cols){
        int index = r*Cols + c;
        d_w[index] = eta * d_dotP[index] + alpha * d_w[index];
    }

}

__global__ void outputError(float* d_error, float t, float* out_layer, int Rows, int Cols){
    /*
        d_error   -- delta_k
        targets    -- one hot encode 1D array containing 0.9 for target label
        out_layer -- the squashed activations for the output layer
        Rows      -- should be 1 as they are all 1D arrays
        Cols      -- should be the number of ouput nodes 
    */
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(r < Rows && c < Cols){ 
        int index = r*Cols + c;
        if(t == index)
            // 2x10               2x10                    2x10            1          2x10
            d_error[index] = out_layer[index] * (1 - out_layer[index]) * (1 - out_layer[index]);
        else 
            d_error[index] = out_layer[index] * (1 - out_layer[index]) * (0 - out_layer[index]);
    }
    
}


__global__ void hiddenError(float* d_error, float* outputUnits, float* hidden_layer, int Rows, int Cols){
    /*
    d_error       -- delta_j    
    outputUnits   -- the output error dot output weights
    hidden_layer  -- the hidden activations
    Rows          -- should be 1 as they are all 1D arrays
    Cols          -- should be the number of ouput nodes 
    */
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(r < Rows && c < Cols){
        int index = r*Cols + c;
        // 2x10               2x10                      2x10                     1x10
        d_error[index] = hidden_layer[index] * (1 - hidden_layer[index]) * (outputUnits[index]);
    }

}




void error_function(int t, float* z, float* h, float* output_weights, float* delta_k, float* delta_j){
    
    //--------------  DEEIVCE Prep ----------------------
  float *d_z, *d_h, *d_k, *d_j;
  float *outputUnits, *d_outputUnits; 
  int outRows    = BATCH_SIZE,  outCols    = NUM_LABELS;
  int hiddenRows = BATCH_SIZE,  hiddenCols = HIDDEN_SIZE;
  int pRows      = BATCH_SIZE,  pCols = NUM_LABELS;

  
  hipError_t cudaStatus;
  cudaStatus = hipMalloc((void**)&d_z, outRows * outCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_z, z, outRows * outCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);

  cudaStatus = hipMalloc((void**)&d_h, hiddenRows * hiddenCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_h, h, hiddenRows * hiddenCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  
  cudaStatus = hipMalloc((void**)&d_k, outRows * outCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_k, delta_k, outRows * outCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  cudaStatus = hipMalloc((void**)&d_j, hiddenRows * hiddenCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_j, delta_j, hiddenRows * hiddenCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  // used for the dot product of output error and output weights
  cudaStatus = hipMalloc((void**)&d_outputUnits, pRows * pCols * sizeof(float));
  cudaCheckError(cudaStatus);
  
  
  
  // call kernel for weight update for each thread to update a weight
  int blockX = ceil(outRows/BLOCK_WIDTH);
  int blockY = ceil(outCols/BLOCK_WIDTH);
  int threadX = BLOCK_WIDTH;
  int threadY = BLOCK_WIDTH;
  dim3 dimGrid(blockX,   blockY,  1);
  dim3 dimBlock(threadX, threadY, 1);
  //--------------  END: DEEIVCE Prep  ----------------------
  
  
  outputError<<<dimGrid, dimBlock>>>(d_k, t, z, outRows, outCols ); 
  // copy back to the host because we need delta K for the dotP
  cudaStatus = hipMemcpy(delta_k, d_k, BATCH_SIZE * outCols * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError(cudaStatus);
  int delta_kRows = outRows;
  int delta_kCols = outCols;

#ifdef DEBUG   
  printf("IN ERROR \nPrinting OUTPUT Weights:\n");
  printMatrix((float*)output_weights, hiddenCols, outCols);
  printf("\n");
  
  printf("IN ERROR \nPrinting DELTA K:\n");
  printMatrix((float*)delta_k, delta_kRows, delta_kRows);
  printf("\n");
#endif
  
  outputUnits = (float*)malloc(pRows*pCols*sizeof(float));
  // output error dot output weights = outputUnits
  //    2x10 @ 10x10  = 2x10
  dotProduct(delta_k, (float*)output_weights, outputUnits, delta_kRows, delta_kCols, HIDDEN_SIZE, NUM_LABELS);
  
  // Prep for hidden error
  blockX = ceil(hiddenCols/2);
  blockY = ceil(hiddenRows/2);
  threadX = BLOCK_WIDTH;
  threadY = BLOCK_WIDTH;
  dim3 dimGrid2(blockX,   blockY,  1);
  dim3 dimBlock2(threadX, threadY, 1);
  

  cudaStatus = hipMemcpy(d_outputUnits, outputUnits, pRows * pCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);
  
  hiddenError<<<dimGrid2, dimBlock2>>>(d_j, d_outputUnits, d_h, hiddenRows, hiddenCols );
  
  // copy back to the host variables
  cudaStatus = hipMemcpy(delta_j, d_j, sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError(cudaStatus);
  
  // deallocate device memory
  hipFree(d_z);
  hipFree(d_h);
  hipFree(d_k);
  hipFree(d_j);
  hipFree(outputUnits);
    
}
void update_weights(float eta, float alpha, float* hidden_weights, int wRows, int wCols, float* dotP, int pRows, int pCols){
/*
    dotP -- error Transposed @ current layer activations
*/

  //--------------  DEEIVCE Prep ----------------------
  float *d_w,  *d_dotP;

  hipError_t cudaStatus;    
  cudaStatus = hipMalloc((void**)&d_w, wRows * wCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_w, hidden_weights, wRows * wCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);

  cudaStatus = hipMalloc((void**)&d_dotP, pRows * pCols * sizeof(float));
  cudaCheckError(cudaStatus);
  cudaStatus = hipMemcpy(d_dotP, dotP, pRows * pCols * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckError(cudaStatus);


  // call kernel for weight update for each thread to update a weight
  int blockX = ceil((float)wCols / BLOCK_WIDTH);
  int blockY = ceil((float)wRows / BLOCK_WIDTH);
  int threadX = BLOCK_WIDTH;
  int threadY = BLOCK_WIDTH;
  dim3 dimGrid(blockX,   blockY,  1);
  dim3 dimBlock(threadX, threadY, 1);
  //--------------  END: DEEIVCE Prep ----------------------

                          
  // output-hidden    (1x10) hidden activations  DOT  error(1x10)
  // hidden-input     (1x785) inputs  DOT  error(1x10) 
  updateWeights<<<dimGrid, dimBlock>>>(eta, alpha, d_dotP, wRows, wCols, d_w);

  
    // copy back to the host variables
  cudaStatus = hipMemcpy(hidden_weights, d_w,  wRows * wCols * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError(cudaStatus);
  
    
    // deallocate device memory
  hipFree(d_w);
  hipFree(d_dotP);
}
